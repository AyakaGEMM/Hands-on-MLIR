#include "hip/hip_runtime.h"
#include "ExecutionEngine/ExecutionEngine.h"
#include "ExecutionEngine/HandsOnNVGPURunnerUtils.h"
#include "ExecutionEngine/HandsOnRunnerUtils.h"
#include "NVGPUKernels/Utils.h"
#include "mlir/ExecutionEngine/CRunnerUtils.h"
#include "llvm/Support/Error.h"
#include <cstddef>
#include <cstdint>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <vector>

#define RowMajor(A, des, i, j, k)                                              \
  ((A)[(i) * (des).strides[0] + (j) * (des).strides[1] +                       \
       (k) * (des).strides[2]])

int main() {
  constexpr int64_t bs = 2;
  constexpr int64_t seq_len = 64;
  constexpr int64_t output_size = 30522;
  constexpr int64_t real_len = 10;
  auto input_ids =
      allocHelper<int64_t, 2, int64_t>({bs, seq_len}, nvgpuAllocer);
  auto mask = allocHelper<int64_t, 2, int64_t>({bs, seq_len}, nvgpuAllocer);
  auto type_id = allocHelper<int64_t, 2, int64_t>({bs, seq_len}, nvgpuAllocer);

  auto input_ids_des =
      static_cast<StridedMemRefType<int64_t, 2> *>(input_ids.descriptor);
  auto mask_des = static_cast<StridedMemRefType<int64_t, 2> *>(mask.descriptor);
  auto type_id_des =
      static_cast<StridedMemRefType<int64_t, 2> *>(type_id.descriptor);

  std::vector<int64_t> input_ids_data(seq_len * bs);

  std::ifstream in;
  in.open("0.txt");
  int64_t a;
  size_t ii = 0;
  while (in >> a) {
    assert(ii < input_ids_data.size());
    input_ids_data[ii++] = a;
  }
  in.close();

  checkCudaErrors(hipMemcpy(input_ids_des->data, input_ids_data.data(),
                             sizeof(int64_t) * input_ids_data.size(),
                             hipMemcpyHostToDevice));

  in.open("1.txt");
  ii = 0;
  while (in >> a) {
    assert(ii < input_ids_data.size());
    input_ids_data[ii++] = a;
  }
  in.close();

  checkCudaErrors(hipMemcpy(mask_des->data, input_ids_data.data(),
                             sizeof(int64_t) * input_ids_data.size(),
                             hipMemcpyHostToDevice));

  in.open("2.txt");
  ii = 0;
  while (in >> a) {
    assert(ii < input_ids_data.size());
    input_ids_data[ii++] = a;
  }

  checkCudaErrors(hipMemcpy(type_id_des->data, input_ids_data.data(),
                             sizeof(int64_t) * input_ids_data.size(),
                             hipMemcpyHostToDevice));

  UnrankedMemRefType<half> b;
  mlir::hands_on_mlir::ExecutionEngine e("libbert_nvgpu.so");

  // Warm up
  auto res =
      e.invoke("forward", input_ids.rank, input_ids.descriptor, mask.rank,
               mask.descriptor, type_id.rank, type_id.descriptor,
               mlir::hands_on_mlir::ExecutionEngine::result(b));
  if (res) {
    llvm::handleAllErrors(std::move(res));
  }

  for (int i = 0; i < 10; i++) {
    res = e.invoke("forward", input_ids.rank, input_ids.descriptor, mask.rank,
                   mask.descriptor, type_id.rank, type_id.descriptor,
                   mlir::hands_on_mlir::ExecutionEngine::result(b));
    if (res) {
      llvm::handleAllErrors(std::move(res));
    }
  }

  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  checkCudaErrors(hipEventRecord(start));

  for (int i = 0; i < 1000; i++) {
    res = e.invoke("forward", input_ids.rank, input_ids.descriptor, mask.rank,
                   mask.descriptor, type_id.rank, type_id.descriptor,
                   mlir::hands_on_mlir::ExecutionEngine::result(b));
    if (res) {
      llvm::handleAllErrors(std::move(res));
    }
  }

  checkCudaErrors(hipEventRecord(stop));
  checkCudaErrors(hipEventSynchronize(stop));
  float msecTotal = 0;
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

  std::cout << "E2E latency: " << msecTotal / 1000 / 1000 << "s" << std::endl;

  auto c = DynamicMemRefType<half>(b);
  std::cout << c.rank << std::endl;

  std::vector<half> thing;
  in.close();
  in.open("3.txt");
  float bb;
  while (in >> bb) {
    thing.emplace_back(bb);
  }

  half *data = new half[bs * seq_len * output_size];

  checkCudaErrors(hipMemcpy(data, c.data,
                             sizeof(half) * bs * seq_len * output_size,
                             hipMemcpyDeviceToHost));
  for (int i = 0; i < c.sizes[0]; i++) {
    for (int j = 0; j < real_len; j++) {
      for (int k = 0; k < c.sizes[2]; k++) {
        if (std::abs(float(RowMajor(data, c, i, j, k) -
                           RowMajor(thing, c, i, j, k))) > 1e-2) {
          std::cout << "Not ok" << std::endl;
        }
      }
    }
  }

  hipFree(c.data);

  free(input_ids.descriptor);
  free(mask.descriptor);
  free(b.descriptor);
}
